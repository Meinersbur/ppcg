#include <assert.h>
#include <stdio.h>
#include "chemv_kernel.hu"
/*
 * Copyright 2014      ARM Ltd.
 *
 * Use of this software is governed by the MIT license
 */

#include <stdio.h>
#include <stdlib.h>

struct ComplexFloat
{
	float Re;
	float Im;
};

/* chemv - complex hermitian matrix-vector multiplication
 * The function body was taken from a VOBLA-generated BLAS library.
 */
void chemv(int n, float alpha_re, float alpha_im,
	int ldAT, struct ComplexFloat AT[restrict const static n][ldAT],
	int incX, struct ComplexFloat X[restrict const static n][incX],
	float beta_re, float beta_im,
	int incY, struct ComplexFloat Y[restrict const static n][incY])
{
	#define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
	if (n >= 1) {
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)

	  struct ComplexFloat *dev_AT;
	  struct ComplexFloat *dev_X;
	  struct ComplexFloat *dev_Y;
	  
	  cudaCheckReturn(hipMalloc((void **) &dev_AT, (ppcg_min(n, ldAT)) * (ldAT) * sizeof(struct ComplexFloat)));
	  cudaCheckReturn(hipMalloc((void **) &dev_X, (n) * (incX) * sizeof(struct ComplexFloat)));
	  cudaCheckReturn(hipMalloc((void **) &dev_Y, (n) * (incY) * sizeof(struct ComplexFloat)));
	  
	  if (ldAT >= n + 1 || ldAT >= 1)
	    cudaCheckReturn(hipMemcpy(dev_AT, AT, (ppcg_min(n, ldAT)) * (ldAT) * sizeof(struct ComplexFloat), hipMemcpyHostToDevice));
	  if (incX >= 1)
	    cudaCheckReturn(hipMemcpy(dev_X, X, (n) * (incX) * sizeof(struct ComplexFloat), hipMemcpyHostToDevice));
	  if (incY >= 1)
	    cudaCheckReturn(hipMemcpy(dev_Y, Y, (n) * (incY) * sizeof(struct ComplexFloat), hipMemcpyHostToDevice));
	  {
	    dim3 k0_dimBlock(32);
	    dim3 k0_dimGrid(ppcg_min(32768, (n + 31) / 32));
	    kernel0 <<<k0_dimGrid, k0_dimBlock>>> (dev_AT, dev_X, dev_Y, alpha_im, alpha_re, beta_im, beta_re, incY, n, incX, ldAT);
	    cudaCheckKernel();
	  }
	  
	  if (incY >= 1)
	    cudaCheckReturn(hipMemcpy(Y, dev_Y, (n) * (incY) * sizeof(struct ComplexFloat), hipMemcpyDeviceToHost));
	  cudaCheckReturn(hipFree(dev_AT));
	  cudaCheckReturn(hipFree(dev_X));
	  cudaCheckReturn(hipFree(dev_Y));
	}
}

int main()
{
	const int n = 37;
	const int incX = 1;
	const int incY = 1;
	const int ldAT = n;
	struct ComplexFloat AT[n][ldAT];
	struct ComplexFloat X[n][incX];
	struct ComplexFloat Y[n][incY];

	for (int i = 0; i < n; i++) {
		X[i][0] = (struct ComplexFloat){i + 5, i * 2};
		Y[i][0] = (struct ComplexFloat){i * 3, i + 7};
		for (int j = 0; j < ldAT; j++) {
			AT[i][j] = (struct ComplexFloat){i + j, i + 3};
		}
	}

	chemv(n, 3.14f, 1.59f, ldAT, AT, incX, X, 2.71f, 8.28f, incY, Y);

	for (int i = 0; i < n; i++)
		printf("%0.2f %0.2f\n", Y[i][0].Re, Y[i][0].Im);

	return EXIT_SUCCESS;
}
